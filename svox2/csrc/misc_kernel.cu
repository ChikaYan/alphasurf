#include "hip/hip_runtime.h"
// Copyright 2021 Alex Yu
// Miscellaneous kernels (3D mask dilate, weight thresholding)

#include <torch/extension.h>
#include <cstdint>
#include <cstdio>
#include <iostream>
#include "cuda_util.cuh"
#include "render_util.cuh"
#include "data_spec_packed.cuh"
#include "cubemap_util.cuh"

#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

namespace {

const int MISC_CUDA_THREADS = 256;
const int MISC_MIN_BLOCKS_PER_SM = 4;
namespace device {

// Can also implement using convs.....
__launch_bounds__(MISC_CUDA_THREADS, MISC_MIN_BLOCKS_PER_SM)
__global__ void dilate_kernel(
        const torch::PackedTensorAccessor32<bool, 3, torch::RestrictPtrTraits> grid,
        // Output
        torch::PackedTensorAccessor32<bool, 3, torch::RestrictPtrTraits> out_grid) {
    CUDA_GET_THREAD_ID(tid, grid.size(0) * grid.size(1) * grid.size(2));

    const int z = tid % grid.size(2);
    const int xy = tid / grid.size(2);
    const int y = xy % grid.size(1);
    const int x = xy / grid.size(1);

    int xl = max(x - 1, 0), xr = min(x + 1, (int) grid.size(0) - 1);
    int yl = max(y - 1, 0), yr = min(y + 1, (int) grid.size(1) - 1);
    int zl = max(z - 1, 0), zr = min(z + 1, (int) grid.size(2) - 1);

    out_grid[x][y][z] =
        grid[xl][yl][zl] | grid[xl][yl][z] | grid[xl][yl][zr] |
        grid[xl][y][zl] | grid[xl][y][z] | grid[xl][y][zr] |
        grid[xl][yr][zl] | grid[xl][yr][z] | grid[xl][yr][zr] |

        grid[x][yl][zl] | grid[x][yl][z] | grid[x][yl][zr] |
        grid[x][y][zl] | grid[x][y][z] | grid[x][y][zr] |
        grid[x][yr][zl] | grid[x][yr][z] | grid[x][yr][zr] |

        grid[xr][yl][zl] | grid[xr][yl][z] | grid[xr][yl][zr] |
        grid[xr][y][zl] | grid[xr][y][z] | grid[xr][y][zr] |
        grid[xr][yr][zl] | grid[xr][yr][z] | grid[xr][yr][zr];
}

// Probably can speed up the following functions, however they are really not
// the bottleneck

// ** Distance transforms
// TODO: Maybe replace this with an euclidean distance transform eg PBA
// Actual L-infty distance transform; turns out this is slower than the geometric way
__launch_bounds__(MISC_CUDA_THREADS, MISC_MIN_BLOCKS_PER_SM)
__global__ void accel_linf_dist_transform_kernel(
        torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> grid,
        int32_t* __restrict__ tmp,
        int d2) {
    const int d0 = d2 == 0 ? 1 : 0;
    const int d1 = 3 - d0 - d2;
    CUDA_GET_THREAD_ID(tid, grid.size(d0) * grid.size(d1));
    int32_t* __restrict__ tmp_ptr = tmp + tid * grid.size(d2);
    int l[3];
    l[d0] = tid / grid.size(1);
    l[d1] = tid % grid.size(1);
    l[d2] = 0;
    const int INF = 0x3f3f3f3f;
    int d01_dist = INF;
    int d2_dist = INF;
    for (; l[d2] < grid.size(d2); ++l[d2]) {
        const int val = grid[l[0]][l[1]][l[2]];
        int32_t cdist = max(- val, 0);
        if (d2 == 0 && cdist)
            cdist = INF;
        const int32_t pdist = max(d2_dist, d01_dist);

        if (cdist < pdist) {
            d01_dist = cdist;
            d2_dist = 0;
        }
        tmp_ptr[l[d2]] = min(cdist, pdist);
        ++d2_dist;
    }

    l[d2] = grid.size(d2) - 1;
    d01_dist = INF;
    d2_dist = INF;
    for (; l[d2] >= 0; --l[d2]) {
        const int val = grid[l[0]][l[1]][l[2]];
        int32_t cdist = max(- val, 0);
        if (d2 == 0 && cdist)
            cdist = INF;
        const int32_t pdist = max(d2_dist, d01_dist);

        if (cdist < pdist) {
            d01_dist = cdist;
            d2_dist = 0;
        }
        if (cdist) {
            grid[l[0]][l[1]][l[2]] = -min(tmp_ptr[l[d2]], min(cdist, pdist));
        }
        ++d2_dist;
    }
}

// Geometric L-infty distance transform-ish thing
__launch_bounds__(MISC_CUDA_THREADS, MISC_MIN_BLOCKS_PER_SM)
__global__ void accel_dist_set_kernel(
        const torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> grid,
        bool* __restrict__ tmp) {
    int sz_x = grid.size(0);
    int sz_y = grid.size(1);
    int sz_z = grid.size(2);
    CUDA_GET_THREAD_ID(tid, sz_x * sz_y * sz_z);

    int z = tid % grid.size(2);
    const int xy = tid / grid.size(2);
    int y = xy % grid.size(1);
    int x = xy / grid.size(1);

    bool* tmp_base = tmp;

    if (grid[x][y][z] >= 0) {
        while (sz_x > 1 && sz_y > 1 && sz_z > 1) {
            // Propagate occupied cell throughout the temp tree parent nodes
            x >>= 1;
            y >>= 1;
            z >>= 1;
            sz_x = int_div2_ceil(sz_x);
            sz_y = int_div2_ceil(sz_y);
            sz_z = int_div2_ceil(sz_z);

            const int idx = x * sz_y * sz_z + y * sz_z + z;
            // printf("s %d  %d %d %d  %d\n", tid, x, y, z, idx);
            tmp_base[idx] = true;
            tmp_base += sz_x * sz_y * sz_z;
        }
    }
}

__launch_bounds__(MISC_CUDA_THREADS, MISC_MIN_BLOCKS_PER_SM)
__global__ void accel_dist_prop_kernel(
        torch::PackedTensorAccessor32<int32_t, 3, torch::RestrictPtrTraits> grid,
        const bool* __restrict__ tmp) {
    int sz_x = grid.size(0);
    int sz_y = grid.size(1);
    int sz_z = grid.size(2);
    CUDA_GET_THREAD_ID(tid, sz_x * sz_y * sz_z);

    int z = tid % grid.size(2);
    const int xy = tid / grid.size(2);
    int y = xy % grid.size(1);
    int x = xy / grid.size(1);
    const bool* tmp_base = tmp;
    int32_t* __restrict__ val = &grid[x][y][z];

    if (*val < 0) {
        int result = -1;
        while (sz_x > 1 && sz_y > 1 && sz_z > 1) {
            // Find the lowest set parent cell if it exists
            x >>= 1;
            y >>= 1;
            z >>= 1;
            sz_x = int_div2_ceil(sz_x);
            sz_y = int_div2_ceil(sz_y);
            sz_z = int_div2_ceil(sz_z);

            const int idx = x * sz_y * sz_z + y * sz_z + z;
            // printf("p %d  %d %d %d  %d tb[%d] , %d %d %d\n", tid, x, y, z, idx, tmp_base[idx],
            //         sz_x, sz_y, sz_z);
            if (tmp_base[idx]) {
                break;
            }
            result -= 1;
            tmp_base += sz_x * sz_y * sz_z;
        }
        *val = result;
    }
}

// Fast single-channel rendering for weight-thresholding
__device__ __inline__ void grid_trace_ray(
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits>
        data,
        SingleRaySpec ray,
        const float* __restrict__ offset,
        const float* __restrict__ scaling,
        float step_size,
        float stop_thresh,
        bool last_sample_opaque,
    torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits>
        grid_weight) {

    // Warning: modifies ray.origin
    transform_coord(ray.origin, scaling, offset);
    // Warning: modifies ray.dir
    const float world_step = _get_delta_scale(scaling, ray.dir) * step_size;

    float t, tmax;
    {
        float t1, t2;
        t = 0.0f;
        tmax = 2e3f;
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            const float invdir = 1.0 / ray.dir[i];
            t1 = (-0.5f - ray.origin[i]) * invdir;
            t2 = (data.size(i) - 0.5f  - ray.origin[i]) * invdir;
            if (ray.dir[i] != 0.f) {
                t = max(t, min(t1, t2));
                tmax = min(tmax, max(t1, t2));
            }
        }
    }

    if (t > tmax) {
        // Ray doesn't hit box
        return;
    }
    float pos[3];
    int32_t l[3];

    float log_light_intensity = 0.f;
    const int stride0 = data.size(1) * data.size(2);
    const int stride1 = data.size(2);
    while (t <= tmax) {
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            pos[j] = ray.origin[j] + t * ray.dir[j];
            pos[j] = min(max(pos[j], 0.f), data.size(j) - 1.f);
            l[j] = (int32_t) pos[j];
            l[j] = min(l[j], data.size(j) - 2);
            pos[j] -= l[j];
        }

        float log_att;
        const int idx = l[0] * stride0 + l[1] * stride1 + l[2];

        float sigma;
        {
            // Trilerp
            const float* __restrict__ sigma000 = data.data() + idx;
            const float* __restrict__ sigma100 = sigma000 + stride0;
            const float ix0y0 = lerp(sigma000[0], sigma000[1], pos[2]);
            const float ix0y1 = lerp(sigma000[stride1], sigma000[stride1 + 1], pos[2]);
            const float ix1y0 = lerp(sigma100[0], sigma100[1], pos[2]);
            const float ix1y1 = lerp(sigma100[stride1], sigma100[stride1 + 1], pos[2]);
            const float ix0 = lerp(ix0y0, ix0y1, pos[1]);
            const float ix1 = lerp(ix1y0, ix1y1, pos[1]);
            sigma = lerp(ix0, ix1, pos[0]);
        }
        if (last_sample_opaque && t + step_size > tmax) {
            sigma += 1e9;
            log_light_intensity = 0.f;
        }

        if (sigma > 1e-8f) {
            log_att = -world_step * sigma;
            const float weight = _EXP(log_light_intensity) * (1.f - _EXP(log_att));
            log_light_intensity += log_att;
            float* __restrict__ max_wt_ptr_000 = grid_weight.data() + idx;
            atomicMax(max_wt_ptr_000, weight);
            atomicMax(max_wt_ptr_000 + 1, weight);
            atomicMax(max_wt_ptr_000 + stride1, weight);
            atomicMax(max_wt_ptr_000 + stride1 + 1, weight);
            float* __restrict__ max_wt_ptr_100 = max_wt_ptr_000 + stride0;
            atomicMax(max_wt_ptr_100, weight);
            atomicMax(max_wt_ptr_100 + 1, weight);
            atomicMax(max_wt_ptr_100 + stride1, weight);
            atomicMax(max_wt_ptr_100 + stride1 + 1, weight);

            if (_EXP(log_light_intensity) < stop_thresh) {
                break;
            }
        }
        t += step_size;
    }
}


// Fast single-channel rendering for weight-thresholding
__device__ __inline__ void sprase_grid_trace_ray(
        const PackedSparseGridSpec& __restrict__ grid,
        SingleRaySpec ray,
        const float* __restrict__ offset,
        const float* __restrict__ scaling,
        float step_size,
        float stop_thresh,
    torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits>
        grid_weight) {

    // Warning: modifies ray.origin
    transform_coord(ray.origin, scaling, offset);
    // Warning: modifies ray.dir
    const float world_step = _get_delta_scale(scaling, ray.dir) * step_size;

    float t, tmax;
    {
        float t1, t2;
        t = 0.0f;
        tmax = 2e3f;
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            const float invdir = 1.0 / ray.dir[i];
            t1 = (-0.5f - ray.origin[i]) * invdir;
            t2 = (grid.size[i] - 0.5f  - ray.origin[i]) * invdir;
            if (ray.dir[i] != 0.f) {
                t = max(t, min(t1, t2));
                tmax = min(tmax, max(t1, t2));
            }
        }
    }

    if (t > tmax) {
        // Ray doesn't hit box
        return;
    }
    float pos[3];
    int32_t l[3];

    float log_light_intensity = 0.f;
    const int stride0 = grid.size[1] * grid.size[2];
    const int stride1 = grid.size[2];
    while (t <= tmax) {
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            pos[j] = ray.origin[j] + t * ray.dir[j];
            pos[j] = min(max(pos[j], 0.f), grid.size[j] - 1.f);
            l[j] = (int32_t) pos[j];
            l[j] = min(l[j], grid.size[j] - 2);
            pos[j] -= l[j];
        }

        // const float skip = compute_skip_dist(ray,
        //                grid.links, grid.stride_x,
        //                grid.size[2], 0);

        // if (skip >= step_size) {
        //     // For consistency, we skip the by step size
        //     t += ceilf(skip / step_size) * step_size;
        //     continue;
        // }

        float log_att;
        const int idx = l[0] * stride0 + l[1] * stride1 + l[2];

        float sigma = trilerp_cuvol_one(
                grid.links, grid.density_data,
                grid.stride_x,
                grid.size[2],
                1,
                l, pos,
                0);
        // printf("sigma: %f\n", sigma);

        // float sigma;
        // {
        //     // Trilerp
        //     const float* __restrict__ sigma000 = data.data() + idx;
        //     const float* __restrict__ sigma100 = sigma000 + stride0;
        //     const float ix0y0 = lerp(sigma000[0], sigma000[1], pos[2]);
        //     const float ix0y1 = lerp(sigma000[stride1], sigma000[stride1 + 1], pos[2]);
        //     const float ix1y0 = lerp(sigma100[0], sigma100[1], pos[2]);
        //     const float ix1y1 = lerp(sigma100[stride1], sigma100[stride1 + 1], pos[2]);
        //     const float ix0 = lerp(ix0y0, ix0y1, pos[1]);
        //     const float ix1 = lerp(ix1y0, ix1y1, pos[1]);
        //     sigma = lerp(ix0, ix1, pos[0]);
        // }
        // if (last_sample_opaque && t + step_size > tmax) {
        //     sigma += 1e9;
        //     log_light_intensity = 0.f;
        // }

        if (sigma > 1e-8f) {
            log_att = -world_step * sigma;
            const float weight = _EXP(log_light_intensity) * (1.f - _EXP(log_att));
            float* __restrict__ max_wt_ptr_000 = grid_weight.data() + idx;
            atomicMax(max_wt_ptr_000, _EXP(log_light_intensity));
            atomicMax(max_wt_ptr_000 + 1, _EXP(log_light_intensity));
            atomicMax(max_wt_ptr_000 + stride1, _EXP(log_light_intensity));
            atomicMax(max_wt_ptr_000 + stride1 + 1, _EXP(log_light_intensity));
            float* __restrict__ max_wt_ptr_100 = max_wt_ptr_000 + stride0;
            atomicMax(max_wt_ptr_100, _EXP(log_light_intensity));
            atomicMax(max_wt_ptr_100 + 1, _EXP(log_light_intensity));
            atomicMax(max_wt_ptr_100 + stride1, _EXP(log_light_intensity));
            atomicMax(max_wt_ptr_100 + stride1 + 1, _EXP(log_light_intensity));

            log_light_intensity += log_att;
            if (_EXP(log_light_intensity) < stop_thresh) {
                break;
            }
        }
        t += step_size;
    }
}

// Fast single-channel rendering for weight-thresholding
__device__ __inline__ void sprase_grid_mask_trace_ray(
        const PackedSparseGridSpec& __restrict__ grid,
        SingleRaySpec ray,
        float near_clip,
        // float stop_thresh,
        float* __restrict__ grid_mask) {

    float step_size = 0.1;
    float stop_thresh = 0.f; 

    // Warning: modifies ray.origin
    transform_coord(ray.origin, grid._scaling, grid._offset);
    // Warning: modifies ray.dir
    const float world_step = _get_delta_scale(grid._scaling, ray.dir) * step_size;

    float t, tmax;
    {
        float t1, t2;
        t = 0.0f;
        tmax = 2e3f;
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            const float invdir = 1.0 / ray.dir[i];
            t1 = (-0.5f - ray.origin[i]) * invdir;
            t2 = (grid.size[i] - 0.5f  - ray.origin[i]) * invdir;
            if (ray.dir[i] != 0.f) {
                t = max(t, min(t1, t2));
                tmax = min(tmax, max(t1, t2));
            }
        }
    }
    if (t < near_clip) t = near_clip;

    if (t > tmax) {
        // Ray doesn't hit box
        return;
    }
    float pos[3];
    int32_t l[3];

    // float log_light_intensity = 0.f;
    const int stride0 = grid.size[1] * grid.size[2];
    const int stride1 = grid.size[2];
    while (t <= tmax) {
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            pos[j] = ray.origin[j] + t * ray.dir[j];
            pos[j] = min(max(pos[j], 0.f), grid.size[j] - 1.f);
            l[j] = (int32_t) pos[j];
            l[j] = min(l[j], grid.size[j] - 2);
            pos[j] -= l[j];
        }

        // float log_att;
        const int idx = l[0] * stride0 + l[1] * stride1 + l[2];

        // float sigma;
        // {
        //     // Trilerp
        //     const float* __restrict__ sigma000 = grid.density_data + idx;
        //     const float* __restrict__ sigma100 = sigma000 + stride0;
        //     const float ix0y0 = lerp(sigma000[0], sigma000[1], pos[2]);
        //     const float ix0y1 = lerp(sigma000[stride1], sigma000[stride1 + 1], pos[2]);
        //     const float ix1y0 = lerp(sigma100[0], sigma100[1], pos[2]);
        //     const float ix1y1 = lerp(sigma100[stride1], sigma100[stride1 + 1], pos[2]);
        //     const float ix0 = lerp(ix0y0, ix0y1, pos[1]);
        //     const float ix1 = lerp(ix1y0, ix1y1, pos[1]);
        //     sigma = lerp(ix0, ix1, pos[0]);
        // }

        const int32_t* __restrict__ link_ptr = grid.links + idx;

        #define MAYBE_ASSIGN_MASK(u) if (link_ptr[u] >= 0) atomicMax(grid_mask + link_ptr[u], 1.f)
        MAYBE_ASSIGN_MASK(0);
        MAYBE_ASSIGN_MASK(1);
        MAYBE_ASSIGN_MASK(stride1);
        MAYBE_ASSIGN_MASK(stride1+1);
        MAYBE_ASSIGN_MASK(stride0);
        MAYBE_ASSIGN_MASK(stride0+1);
        MAYBE_ASSIGN_MASK(stride0+stride1);
        MAYBE_ASSIGN_MASK(stride0+stride1+1);
        #undef MAYBE_ASSIGN_MASK

        // if (sigma > 1e-8f) {
        //     log_att = -world_step * sigma;
        //     const float weight = _EXP(log_light_intensity) * (1.f - _EXP(log_att));
        //     log_light_intensity += log_att;
        //     // float* __restrict__ max_wt_ptr_000 = grid_weight.data() + idx;
        //     // atomicMax(max_wt_ptr_000, weight);
        //     // atomicMax(max_wt_ptr_000 + 1, weight);
        //     // atomicMax(max_wt_ptr_000 + stride1, weight);
        //     // atomicMax(max_wt_ptr_000 + stride1 + 1, weight);
        //     // float* __restrict__ max_wt_ptr_100 = max_wt_ptr_000 + stride0;
        //     // atomicMax(max_wt_ptr_100, weight);
        //     // atomicMax(max_wt_ptr_100 + 1, weight);
        //     // atomicMax(max_wt_ptr_100 + stride1, weight);
        //     // atomicMax(max_wt_ptr_100 + stride1 + 1, weight);

        //     if (_EXP(log_light_intensity) < stop_thresh) {
        //         break;
        //     }
        // }
        t += step_size;
    }
}


// Fast single-channel rendering for weight-thresholding
__device__ __inline__ void sparse_grid_visbility_trace_ray_surf(
        const PackedSparseGridSpec& __restrict__ grid,
        SingleRaySpec ray,
        float* __restrict__ visibility_out) {

    // Warning: modifies ray.origin
    transform_coord(ray.origin, grid._scaling, grid._offset);
    // Warning: modifies ray.dir
    _get_delta_scale(grid._scaling, ray.dir);

    float t, tmax;
    {
        float t1, t2;
        t = 0.0f;
        tmax = 2e3f;
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            const float invdir = 1.0 / ray.dir[i];
            t1 = (-0.5f - ray.origin[i]) * invdir;
            t2 = (grid.size[i] - 0.5f  - ray.origin[i]) * invdir;
            if (ray.dir[i] != 0.f) {
                t = max(t, min(t1, t2));
                tmax = min(tmax, max(t1, t2));
            }
        }
    }

    // printf("t, tmax: [%f, %f]\n", t, tmax);
    if (t > tmax) {
        // Ray doesn't hit box
        return;
    }
    float pos[3];
    int32_t l[3];

    float log_light_intensity = 0.f;
    const int stride0 = grid.size[1] * grid.size[2];
    const int stride1 = grid.size[2];

    double const  ray_dir_d[] = {ray.dir[0], ray.dir[1], ray.dir[2]};

    int32_t voxel_l[3];
    int32_t next_voxel[3];
#pragma unroll 3
    for (int j = 0; j < 3; ++j) {
        next_voxel[j] = static_cast<int32_t>(fmaf(t, ray.dir[j], ray.origin[j])); // fmaf(x,y,z) = (x*y)+z
        next_voxel[j] = min(max(next_voxel[j], 0), grid.size[j] - 2);
    }

    // float visibility = 1.f;


    while (t <= tmax) {

        voxel_l[0] = next_voxel[0];
        voxel_l[1] = next_voxel[1];
        voxel_l[2] = next_voxel[2];
        // printf("voxel: [%d, %d, %d]\n", voxel_l[0], voxel_l[1], voxel_l[2]);

        // assign visibility to current voxel

        int const offx = grid.stride_x, offy = grid.size[2];
        const int32_t* __restrict__ link_ptr = grid.links + (offx * voxel_l[0] + offy * voxel_l[1] + voxel_l[2]);

        // if (link_ptr[0] >=0 ){
        //     printf("voxel: [%d, %d, %d]\n", voxel_l[0], voxel_l[1], voxel_l[2]);
        //     atomicMax(visibility_out+ link_ptr[0], visibility);
        // }

        // #define MAYBE_ASSIGN_VIS(u) if (link_ptr[u] >= 0) atomicMax(visibility_out+ link_ptr[u], visibility)
        #define MAYBE_ASSIGN_VIS(u) if (link_ptr[u] >= 0) atomicAdd(visibility_out+ link_ptr[u], 1.f)
        MAYBE_ASSIGN_VIS(0);
        MAYBE_ASSIGN_VIS(1);
        MAYBE_ASSIGN_VIS(stride1);
        MAYBE_ASSIGN_VIS(stride1+1);
        MAYBE_ASSIGN_VIS(stride0);
        MAYBE_ASSIGN_VIS(stride0+1);
        MAYBE_ASSIGN_VIS(stride0+stride1);
        MAYBE_ASSIGN_VIS(stride0+stride1+1);
        #undef MAYBE_ASSIGN_VIS



        // Find close and far intersections between ray and voxel
        int32_t const close_plane[] = {
            ray.dir[0] > 0.f ? voxel_l[0] : voxel_l[0]+1,
            ray.dir[1] > 0.f ? voxel_l[1] : voxel_l[1]+1,
            ray.dir[2] > 0.f ? voxel_l[2] : voxel_l[2]+1,
        };
        int32_t const far_plane[] = {
            ray.dir[0] > 0.f ? voxel_l[0]+1 : voxel_l[0],
            ray.dir[1] > 0.f ? voxel_l[1]+1 : voxel_l[1],
            ray.dir[2] > 0.f ? voxel_l[2]+1 : voxel_l[2],
        };

        // threshold t_close by 0.f to prevent cases where camera origin is within the voxel
        float const t_close = max(max(
            max((static_cast<float>(close_plane[0])-ray.origin[0])/ray.dir[0], (static_cast<float>(close_plane[1])-ray.origin[1])/ray.dir[1]),
            (static_cast<float>(close_plane[2])-ray.origin[2])/ray.dir[2]), 0.f);
        
        float const t_fars [] = {
            (static_cast<float>(far_plane[0])-ray.origin[0])/ray.dir[0],
            (static_cast<float>(far_plane[1])-ray.origin[1])/ray.dir[1],
            (static_cast<float>(far_plane[2])-ray.origin[2])/ray.dir[2]
            };

        float const t_far = min(min(t_fars[0], t_fars[1]), t_fars[2]);

        t = t_far;

        if (t_far == t_fars[0]){
            next_voxel[0] += (ray.dir[0] > 0.f) ?  1 : -1;
            if ((next_voxel[0] < 0) || (next_voxel[0] >= grid.size[0]-1)) t = ray.tmax + 1.f;
        }else if (t_far == t_fars[1]){
            next_voxel[1] += (ray.dir[1] > 0.f) ?  1 : -1;
            if ((next_voxel[1] < 0) || (next_voxel[1] >= grid.size[1]-1)) t = ray.tmax + 1.f;
        }else{
            next_voxel[2] += (ray.dir[2] > 0.f) ?  1 : -1;
            if ((next_voxel[2] < 0) || (next_voxel[2] >= grid.size[2]-1)) t = ray.tmax + 1.f;
        }


        // skip voxel if any of the vertices is turned off
        if ((voxel_l[0] + 1 >= grid.size[0]) || (voxel_l[1] + 1 >= grid.size[1]) || (voxel_l[2] + 1 >= grid.size[2]) \
            || (link_ptr[0] < 0) || (link_ptr[1] < 0) || (link_ptr[offy] < 0) || (link_ptr[offy+1] < 0) \
            || (link_ptr[offx] < 0) || (link_ptr[offx+1] < 0) || (link_ptr[offx+offy] < 0) || (link_ptr[offx+offy+1] < 0)
        ){
            continue;
        }


        double const new_origin[] = {ray.origin[0] + t_close*ray.dir[0], ray.origin[1] + t_close*ray.dir[1], ray.origin[2] + t_close*ray.dir[2]};

        // find intersections
        double const surface[8] = {
            grid.surface_data[link_ptr[0]],
            grid.surface_data[link_ptr[1]],
            grid.surface_data[link_ptr[offy]],
            grid.surface_data[link_ptr[offy+1]],
            grid.surface_data[link_ptr[offx]],
            grid.surface_data[link_ptr[offx+1]],
            grid.surface_data[link_ptr[offx+offy]],
            grid.surface_data[link_ptr[offx+offy+1]],
        };

        double fs[4];
        double const new_norm_origin[] = {new_origin[0] - voxel_l[0], new_origin[1] - voxel_l[1], new_origin[2] - voxel_l[2]};
        // surface_to_cubic_equation(surface, new_origin, ray_dir_d, voxel_l, fs);
        surface_to_cubic_equation_01(surface, new_norm_origin, ray_dir_d, fs);

        const auto mnmax = thrust::minmax_element(thrust::device, surface, surface+8);
        for (int i=0; i < grid.level_set_num; ++i){
            double const lv_set = grid.level_set_data[i];
            if ((lv_set < *mnmax.first) || (lv_set > *mnmax.second)){
                continue;
            }

            ////////////// CUBIC ROOT SOLVING //////////////
            double st[3] = {-1, -1, -1}; // sample t
            // note that it's now distance from new origin to intersections

            cubic_equation_solver_vieta(
                fs[0] - lv_set, fs[1], fs[2], fs[3],
                1e-8, // float eps
                1e-10, // double eps
                st
                );


            ////////////// TRILINEAR INTERPOLATE //////////////
            for (int j=0; j < 3; ++j){
                if (st[j] <= 0){
                    // ignore intersection at negative direction
                    continue;
                }

#pragma unroll 3
                for (int k=0; k < 3; ++k){
                    // assert(!isnan(st[j]));
                    ray.pos[k] = fmaf(static_cast<float>(st[j]), ray.dir[k], static_cast<float>(new_origin[k])); // fmaf(x,y,z) = (x*y)+z
                    ray.l[k] = min(voxel_l[k], grid.size[k] - 2); // get l
                    ray.pos[k] -= static_cast<float>(ray.l[k]); // get trilinear interpolate distances
                }

                // check if intersection is within grid
                if ((ray.pos[0] < 0) | (ray.pos[0] > 1) | (ray.pos[1] < 0) | (ray.pos[1] > 1) | (ray.pos[2] < 0) | (ray.pos[2] > 1)){
                    continue;
                }

                // vox_has_sample = true;
                // float alpha = trilerp_cuvol_one(
                //         grid.links, grid.density_data,
                //         grid.stride_x,
                //         grid.size[2],
                //         1,
                //         ray.l, ray.pos,
                //         0);

                // visibility = 0.f;

                return;


            }

        }
    }
}

// Fast single-channel rendering for surface weight-thresholding
__device__ __inline__ void grid_trace_ray_surface(
        const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> density_data,
        const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> surface_data,
        SingleRaySpec ray,
        const float* __restrict__ offset,
        const float* __restrict__ scaling,
        float step_size,
        float stop_thresh,
        torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits> grid_weight) {

    // Warning: modifies ray.origin
    transform_coord(ray.origin, scaling, offset);
    // Warning: modifies ray.dir
    const float world_step = _get_delta_scale(scaling, ray.dir) * step_size;

    double const  ray_dir_d[] = {ray.dir[0], ray.dir[1], ray.dir[2]};
    double const  ray_origin_d[] = {ray.origin[0], ray.origin[1], ray.origin[2]};

    float t, tmax;
    {
        float t1, t2;
        t = 0.0f;
        tmax = 2e3f;
#pragma unroll 3
        for (int i = 0; i < 3; ++i) {
            const float invdir = 1.0 / ray.dir[i];
            t1 = (-0.5f - ray.origin[i]) * invdir;
            t2 = (density_data.size(i) - 0.5f  - ray.origin[i]) * invdir;
            if (ray.dir[i] != 0.f) {
                t = max(t, min(t1, t2));
                tmax = min(tmax, max(t1, t2));
            }
        }
    }

    if (t > tmax) {
        // Ray doesn't hit box
        return;
    }
    float pos[3];
    int32_t voxel_l[3];
    int32_t last_voxel[] = {-1,-1,-1};

    float log_light_intensity = 0.f;
    const int offx = density_data.size(1) * density_data.size(2);
    const int offy = density_data.size(2);
    while (t <= tmax) {
#pragma unroll 3
        for (int j = 0; j < 3; ++j) {
            voxel_l[j] = static_cast<int32_t>(fmaf(t, ray.dir[j], ray.origin[j])); // fmaf(x,y,z) = (x*y)+z
            voxel_l[j] = min(max(voxel_l[j], 0), density_data.size(j) - 2);
        }


        if ((voxel_l[0] == last_voxel[0]) && (voxel_l[1] == last_voxel[1]) && (voxel_l[2] == last_voxel[2])){
            // const float skip = compute_skip_dist(ray,
            //             grid.links, grid.stride_x,
            //             grid.size[2], 0);

            t += step_size;
            continue;
        }


        // const int32_t* __restrict__ link_ptr = grid.links + (offx * voxel_l[0] + offy * voxel_l[1] + voxel_l[2]);

        // // skip voxel if any of the vertices is turned off
        // if ((voxel_l[0] + 1 >= grid.size[0]) || (voxel_l[1] + 1 >= grid.size[1]) || (voxel_l[2] + 1 >= grid.size[2]) \
        //     || (link_ptr[0] < 0) || (link_ptr[1] < 0) || (link_ptr[offy] < 0) || (link_ptr[offy+1] < 0) \
        //     || (link_ptr[offx] < 0) || (link_ptr[offx+1] < 0) || (link_ptr[offx+offy] < 0) || (link_ptr[offx+offy+1] < 0)
        // ){
        //     // const float skip = compute_skip_dist(ray,
        //     //             grid.links, grid.stride_x,
        //     //             grid.size[2], 0);

        //     t += step_size;
        //     continue;
        // }

        // last_voxel[0] = voxel_l[0];
        // last_voxel[1] = voxel_l[1];
        // last_voxel[2] = voxel_l[2];




// #pragma unroll 3
//         for (int j = 0; j < 3; ++j) {
//             pos[j] = ray.origin[j] + t * ray.dir[j];
//             pos[j] = min(max(pos[j], 0.f), density_data.size(j) - 1.f);
//             l[j] = (int32_t) pos[j];
//             l[j] = min(l[j], density_data.size(j) - 2);
//             pos[j] -= l[j];
//         }

//         float log_att;
//         const int idx = l[0] * stride0 + l[1] * stride1 + l[2];

//         float sigma;
//         {
//             // Trilerp
//             const float* __restrict__ sigma000 = density_data.data() + idx;
//             const float* __restrict__ sigma100 = sigma000 + stride0;
//             const float ix0y0 = lerp(sigma000[0], sigma000[1], pos[2]);
//             const float ix0y1 = lerp(sigma000[stride1], sigma000[stride1 + 1], pos[2]);
//             const float ix1y0 = lerp(sigma100[0], sigma100[1], pos[2]);
//             const float ix1y1 = lerp(sigma100[stride1], sigma100[stride1 + 1], pos[2]);
//             const float ix0 = lerp(ix0y0, ix0y1, pos[1]);
//             const float ix1 = lerp(ix1y0, ix1y1, pos[1]);
//             sigma = lerp(ix0, ix1, pos[0]);
//         }
//         if (last_sample_opaque && t + step_size > tmax) {
//             sigma += 1e9;
//             log_light_intensity = 0.f;
//         }

//         if (sigma > 1e-8f) {
//             log_att = -world_step * sigma;
//             const float weight = _EXP(log_light_intensity) * (1.f - _EXP(log_att));
//             log_light_intensity += log_att;
//             float* __restrict__ max_wt_ptr_000 = grid_weight.data() + idx;
//             atomicMax(max_wt_ptr_000, weight);
//             atomicMax(max_wt_ptr_000 + 1, weight);
//             atomicMax(max_wt_ptr_000 + stride1, weight);
//             atomicMax(max_wt_ptr_000 + stride1 + 1, weight);
//             float* __restrict__ max_wt_ptr_100 = max_wt_ptr_000 + stride0;
//             atomicMax(max_wt_ptr_100, weight);
//             atomicMax(max_wt_ptr_100 + 1, weight);
//             atomicMax(max_wt_ptr_100 + stride1, weight);
//             atomicMax(max_wt_ptr_100 + stride1 + 1, weight);

//             if (_EXP(log_light_intensity) < stop_thresh) {
//                 break;
//             }
//         }
        t += step_size;
    }
}

// __global__ void sample_cubemap_kernel(
//     const torch::PackedTensorAccessor32<float, 4, torch::RestrictPtrTraits>
//         cubemap,
//     const torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits>
//         dirs,
//     int Q,
//     bool eac,
//     // Output
//     torch::PackedTensorAccessor32<float, 2, torch::RestrictPtrTraits>
//         result) {
//     CUDA_GET_THREAD_ID(tid, Q);
//
//     const int chnl_id = tid % cubemap.size(3);
//     const int ray_id = tid / cubemap.size(3);
//
//     const int face_reso = cubemap.size(1);
//
//     CubemapCoord coord = dir_to_cubemap_coord(dirs[ray_id].data(), face_reso, eac);
//     CubemapBilerpQuery query = cubemap_build_query(coord, face_reso);
//     result[ray_id][chnl_id] = cubemap_sample(
//             cubemap.data(),
//             query,
//             face_reso,
//             cubemap.size(3),
//             chnl_id);
// }

__launch_bounds__(MISC_CUDA_THREADS, MISC_MIN_BLOCKS_PER_SM)
__global__ void grid_weight_render_kernel(
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits>
        data,
    PackedCameraSpec cam,
    float step_size,
    float stop_thresh,
    bool last_sample_opaque,
    const float* __restrict__ offset,
    const float* __restrict__ scaling,
    torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits>
        grid_weight) {
    CUDA_GET_THREAD_ID(tid, cam.width * cam.height);
    int iy = tid / cam.width, ix = tid % cam.width;
    float dir[3], origin[3];
    cam2world_ray(ix, iy, cam, dir, origin);
    grid_trace_ray(
        data,
        SingleRaySpec(origin, dir),
        offset,
        scaling,
        step_size,
        stop_thresh,
        last_sample_opaque,
        grid_weight);
}

__launch_bounds__(MISC_CUDA_THREADS, MISC_MIN_BLOCKS_PER_SM)
__global__ void sparse_grid_weight_render_kernel(
    PackedSparseGridSpec grid,
    PackedCameraSpec cam,
    float step_size,
    float stop_thresh,
    const float* __restrict__ offset,
    const float* __restrict__ scaling,
    torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits>
        grid_weight) {
    CUDA_GET_THREAD_ID(tid, cam.width * cam.height);
    int iy = tid / cam.width, ix = tid % cam.width;
    float dir[3], origin[3];
    cam2world_ray(ix, iy, cam, dir, origin);
    sprase_grid_trace_ray(
        grid,
        SingleRaySpec(origin, dir),
        offset,
        scaling,
        step_size,
        stop_thresh,
        grid_weight);
}

__launch_bounds__(MISC_CUDA_THREADS, MISC_MIN_BLOCKS_PER_SM)
__global__ void sparse_grid_visbility_render_surf_kernel(
    PackedSparseGridSpec grid,
    PackedCameraSpec cam,
    float* __restrict__ visibility_out) {
    CUDA_GET_THREAD_ID(tid, cam.width * cam.height);
    int iy = tid / cam.width, ix = tid % cam.width;
    float dir[3], origin[3];
    cam2world_ray(ix, iy, cam, dir, origin);
    sparse_grid_visbility_trace_ray_surf(
        grid,
        SingleRaySpec(origin, dir),
        visibility_out);
}

__launch_bounds__(MISC_CUDA_THREADS, MISC_MIN_BLOCKS_PER_SM)
__global__ void sparse_grid_mask_render(
    PackedSparseGridSpec grid,
    PackedRaysSpec rays,
    float near_clip,
    float* __restrict__ grid_mask) {
    CUDA_GET_THREAD_ID(tid, int(rays.origins.size(0)));    
    const int ray_id = tid;

    SingleRaySpec ray_spec;
    ray_spec.set(rays.origins[ray_id].data(),
                 rays.dirs[ray_id].data());

    // cam2world_ray(ix, iy, cam, dir, origin);
    sprase_grid_mask_trace_ray(
        grid,
        ray_spec,
        near_clip,
        grid_mask);
}

__launch_bounds__(MISC_CUDA_THREADS, MISC_MIN_BLOCKS_PER_SM)
__global__ void grid_surface_weight_render_kernel(
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits>
        density_data,
    const torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits>
        surface_data,
    PackedCameraSpec cam,
    float step_size,
    float stop_thresh,
    const float* __restrict__ offset,
    const float* __restrict__ scaling,
    torch::PackedTensorAccessor32<float, 3, torch::RestrictPtrTraits>
        grid_weight) {
    CUDA_GET_THREAD_ID(tid, cam.width * cam.height);
    int iy = tid / cam.width, ix = tid % cam.width;
    float dir[3], origin[3];
    cam2world_ray(ix, iy, cam, dir, origin);
    grid_trace_ray_surface(
        density_data,
        surface_data,
        SingleRaySpec(origin, dir),
        offset,
        scaling,
        step_size,
        stop_thresh,
        grid_weight);
}

}  // namespace device
}  // namespace

torch::Tensor dilate(torch::Tensor grid) {
    DEVICE_GUARD(grid);
    CHECK_INPUT(grid);
    TORCH_CHECK(!grid.is_floating_point());
    TORCH_CHECK(grid.ndimension() == 3);

    int Q = grid.size(0) * grid.size(1) * grid.size(2);

    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, MISC_CUDA_THREADS);
    torch::Tensor result = torch::empty_like(grid);
    device::dilate_kernel<<<blocks, MISC_CUDA_THREADS>>>(
            grid.packed_accessor32<bool, 3, torch::RestrictPtrTraits>(),
            // Output
            result.packed_accessor32<bool, 3, torch::RestrictPtrTraits>());
    return result;
}

void accel_dist_prop(torch::Tensor grid) {
    // Grid here is links array from the sparse grid
    DEVICE_GUARD(grid);
    CHECK_INPUT(grid);
    TORCH_CHECK(!grid.is_floating_point());
    TORCH_CHECK(grid.ndimension() == 3);

    int64_t sz_x = grid.size(0);
    int64_t sz_y = grid.size(1);
    int64_t sz_z = grid.size(2);

    int Q = grid.size(0) * grid.size(1) * grid.size(2);

    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, MISC_CUDA_THREADS);

    int64_t req_size = 0;
    while (sz_x > 1 && sz_y > 1 && sz_z > 1) {
        sz_x = int_div2_ceil(sz_x);
        sz_y = int_div2_ceil(sz_y);
        sz_z = int_div2_ceil(sz_z);
        req_size += sz_x * sz_y * sz_z;
    }

    auto tmp_options = torch::TensorOptions()
                  .dtype(torch::kBool)
                  .layout(torch::kStrided)
                  .device(grid.device())
                  .requires_grad(false);
    torch::Tensor tmp = torch::zeros({req_size}, tmp_options);
    device::accel_dist_set_kernel<<<blocks, MISC_CUDA_THREADS>>>(
            grid.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
            tmp.data_ptr<bool>());

    device::accel_dist_prop_kernel<<<blocks, MISC_CUDA_THREADS>>>(
            grid.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
            tmp.data_ptr<bool>());


    // int32_t* tmp;
    // const int req_size = sz_x * sz_y * sz_z;
    // cuda(Malloc(&tmp, req_size * sizeof(int32_t)));
    // cuda(MemsetAsync(tmp, 0, req_size * sizeof(int32_t)));
    //
    // {
    //     for (int d2 = 0; d2 < 3; ++d2) {
    //         int d0 = d2 == 0 ? 1 : 0;
    //         int d1 = 3 - d0 - d2;
    //         int Q = grid.size(d0) * grid.size(d1);
    //
    //         const int blocks = CUDA_N_BLOCKS_NEEDED(Q, MISC_CUDA_THREADS);
    //
    //         device::accel_linf_dist_transform_kernel<<<blocks, MISC_CUDA_THREADS>>>(
    //                 grid.packed_accessor32<int32_t, 3, torch::RestrictPtrTraits>(),
    //                 tmp,
    //                 d2);
    //     }
    // }

    // cuda(Free(tmp));
    CUDA_CHECK_ERRORS;
}

void grid_weight_render(
    torch::Tensor data, CameraSpec& cam,
    float step_size,
    float stop_thresh,
    bool last_sample_opaque,
    torch::Tensor offset, torch::Tensor scaling,
    torch::Tensor grid_weight_out) {
    DEVICE_GUARD(data);
    CHECK_INPUT(data);
    CHECK_INPUT(offset);
    CHECK_INPUT(scaling);
    CHECK_INPUT(grid_weight_out);
    cam.check();
    const size_t Q = size_t(cam.width) * cam.height;

    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, MISC_CUDA_THREADS);

    device::grid_weight_render_kernel<<<blocks, MISC_CUDA_THREADS>>>(
        data.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
        cam,
        step_size,
        stop_thresh,
        last_sample_opaque,
        offset.data_ptr<float>(),
        scaling.data_ptr<float>(),
        grid_weight_out.packed_accessor32<float, 3, torch::RestrictPtrTraits>());
    CUDA_CHECK_ERRORS;
}

void sparse_grid_weight_render(
    SparseGridSpec& grid, CameraSpec& cam,
    float step_size,
    float stop_thresh,
    torch::Tensor offset, torch::Tensor scaling,
    torch::Tensor grid_weight_out) {
    DEVICE_GUARD(grid.density_data);
    grid.check();
    CHECK_INPUT(offset);
    CHECK_INPUT(scaling);
    CHECK_INPUT(grid_weight_out);
    cam.check();
    const size_t Q = size_t(cam.width) * cam.height;

    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, MISC_CUDA_THREADS);

    device::sparse_grid_weight_render_kernel<<<blocks, MISC_CUDA_THREADS>>>(
        grid,
        cam,
        step_size,
        stop_thresh,
        offset.data_ptr<float>(),
        scaling.data_ptr<float>(),
        grid_weight_out.packed_accessor32<float, 3, torch::RestrictPtrTraits>());
    CUDA_CHECK_ERRORS;
}

void sparse_grid_visbility_render_surf(
    SparseGridSpec& grid, CameraSpec& cam,
    torch::Tensor visibility_out) {
    DEVICE_GUARD(grid.density_data);
    grid.check();
    cam.check();
    const size_t Q = size_t(cam.width) * cam.height;

    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, MISC_CUDA_THREADS);

    device::sparse_grid_visbility_render_surf_kernel<<<blocks, MISC_CUDA_THREADS>>>(
        grid,
        cam,
        visibility_out.data_ptr<float>()
    );
    CUDA_CHECK_ERRORS;
}

void sparse_grid_mask_render(
    SparseGridSpec& grid, RaysSpec& rays, float near_clip,
    torch::Tensor grid_mask) {
    DEVICE_GUARD(grid.density_data);
    grid.check();
    rays.check();
    const auto Q = rays.origins.size(0);

    const int blocks = CUDA_N_BLOCKS_NEEDED(Q, MISC_CUDA_THREADS);

    device::sparse_grid_mask_render<<<blocks, MISC_CUDA_THREADS>>>(
        grid,
        rays,
        near_clip,
        grid_mask.data_ptr<float>()
    );
    CUDA_CHECK_ERRORS;
}

// void grid_surface_weight_render(
//     torch::Tensor density_data, 
//     torch::Tensor surface_data, 
//     CameraSpec& cam,
//     float step_size,
//     float stop_thresh,
//     torch::Tensor offset, torch::Tensor scaling,
//     torch::Tensor grid_weight_out) {
//     DEVICE_GUARD(density_data);
//     DEVICE_GUARD(surface_data);
//     CHECK_INPUT(density_data);
//     CHECK_INPUT(surface_data);
//     CHECK_INPUT(offset);
//     CHECK_INPUT(scaling);
//     CHECK_INPUT(grid_weight_out);
//     cam.check();
//     const size_t Q = size_t(cam.width) * cam.height;

//     const int blocks = CUDA_N_BLOCKS_NEEDED(Q, MISC_CUDA_THREADS);

//     device::grid_surface_weight_render_kernel<<<blocks, MISC_CUDA_THREADS>>>(
//         density_data.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
//         surface_data.packed_accessor32<float, 3, torch::RestrictPtrTraits>(),
//         cam,
//         step_size,
//         stop_thresh,
//         offset.data_ptr<float>(),
//         scaling.data_ptr<float>(),
//         grid_weight_out.packed_accessor32<float, 3, torch::RestrictPtrTraits>());
//     CUDA_CHECK_ERRORS;
// }

// For debugging
// void sample_cubemap(torch::Tensor cubemap, // (6, R, R, C)
//                     torch::Tensor dirs,
//                     bool eac,
//                     // Output
//                     torch::Tensor result) {
//     DEVICE_GUARD(cubemap);
//     CHECK_INPUT(cubemap);
//     CHECK_INPUT(dirs);
//     CHECK_INPUT(result);
//     TORCH_CHECK(cubemap.ndimension() == 4);
//     TORCH_CHECK(cubemap.size(0) == 6);
//     TORCH_CHECK(cubemap.size(1) == cubemap.size(2));
//
//     const size_t Q = size_t(dirs.size(0)) * cubemap.size(3);
//     const int blocks = CUDA_N_BLOCKS_NEEDED(Q, MISC_CUDA_THREADS);
//
//     device::sample_cubemap_kernel<<<blocks, MISC_CUDA_THREADS>>>(
//         cubemap.packed_accessor32<float, 4, torch::RestrictPtrTraits>(),
//         dirs.packed_accessor32<float, 2, torch::RestrictPtrTraits>(),
//         Q,
//         eac,
//         // Output
//         result.packed_accessor32<float, 2, torch::RestrictPtrTraits>());
//     CUDA_CHECK_ERRORS;
// }
